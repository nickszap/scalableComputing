/*------------------------------------------------------

         Template taken from vecadd

--------------------------------------------------------*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

// Declare error and timing utilities =========================================

#define FATAL(msg, ...) \
    do {\
        fprintf(stderr, "[%s:%d] "msg"\n\n", __FILE__, __LINE__, ##__VA_ARGS__);\
        exit(-1);\
    } while(0)


typedef struct {
    struct timeval startTime;
    struct timeval endTime;
} Timer;
void startTime(Timer* timer);
void stopTime(Timer* timer);
float elapsedTime(Timer timer);

// cuda kernel ====================================================
const unsigned int nAssets=2; //if at runtime, difficult to handle memory for each thread

#include <hiprand/hiprand_kernel.h>

__host__ __device__ int indexFlat(int iSimu, int iAsset, int nSimu){
  return iAsset*nSimu+iSimu; //neighboring threads should be neighbors in memory
}

__device__ float priceModel(float val, int i){
	return (float) i;
}

__global__ void simPrice(float* prices, float* initialPrices, const int nSimu, const int nSteps) {

    int iSimu = blockDim.x * blockIdx.x + threadIdx.x;
    int iStep, iAsset;
    
    if (iSimu<nSimu){
    	float prices_t[nAssets];
    
    	//initial prices - privatized
    	for (iAsset=0; iAsset<nAssets; iAsset++){
    		prices_t[iAsset] = initialPrices[iAsset];
    	}
    	
    	//time evolution
    	for (iStep=0; iStep<nSteps; iStep++){
    		for (iAsset=0; iAsset<nAssets; iAsset++){
    			prices_t[iAsset] = priceModel(prices_t[iAsset], iSimu);
    		}
    	}
    	
    	//store result
    	for (iAsset=0; iAsset<nAssets; iAsset++){
    		prices[indexFlat(iSimu, iAsset,nSimu)] = prices_t[iAsset];
    	}
    	
    }//iSimu

}

// Main function ==============================================================
int main(int argc, char**argv) {

    Timer timer;
    hipError_t cuda_ret;

    // Initialize GPU/CUDA ----------------------------------------------------

    printf("\nInitializing GPU/CUDA..."); fflush(stdout);
    startTime(&timer);
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));


    // Input parameters and host variables -------------------------

    printf("Setting up the problem..."); fflush(stdout);
    startTime(&timer);

    unsigned int nSimu, nSteps;
    nSimu = 1000;
    nSteps = 100;
    
    unsigned int nTotal = nAssets*nSimu;
    float* prices_h = (float*) malloc( sizeof(float)*nTotal );
    
    float initialPrices_h[nAssets];
    int i;
    for (i=0; i<nAssets; i++){
    	initialPrices_h[i] = 0.0;
    }
    	

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Case = %u assets with %u simulations \n", nAssets, nSimu);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    float* prices_d;
    cuda_ret = hipMalloc((void**) &prices_d, sizeof(float)*nTotal);
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    
    float* initialPrices_d;
    cuda_ret = hipMalloc((void**) &initialPrices_d, sizeof(float)*nAssets);
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------
    
    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMemcpy(initialPrices_d, initialPrices_h, sizeof(float)*nAssets, hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    
    // Launch kernel ----------------------------------------------------------

    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    const unsigned int THREADS_PER_BLOCK = 64;
    const unsigned int numBlocks = (nTotal - 1)/THREADS_PER_BLOCK + 1;
    dim3 gridDim(numBlocks, 1, 1), blockDim(THREADS_PER_BLOCK, 1, 1);
    simPrice <<< gridDim, blockDim >>> (prices_d, initialPrices_d, nSimu,nSteps);

    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMemcpy(prices_h, prices_d, sizeof(float)*nTotal, hipMemcpyDeviceToHost);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory from device");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n\n", elapsedTime(timer));
    
    // print some output ----------------------------------------
    int iAsset, iSimu, iPrice;
    for (iAsset=0; iAsset<nAssets; iAsset++){
    	printf("\nAsset %d price0 %g\n",iAsset, initialPrices_h[iAsset]);
    	for (iSimu=0; iSimu<nSimu; iSimu++){
    		iPrice = indexFlat(iSimu, iAsset,nSimu);
    		printf(" %g ",prices_h[iPrice]);
    	}
    }
    printf("\n");
    
    // Free memory ------------------------------------------------------------

    free(prices_h);

    hipFree(prices_d);

    return 0;

}

// Define timing utilities ====================================================

void startTime(Timer* timer) {
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer* timer) {
    gettimeofday(&(timer->endTime), NULL);
}

float elapsedTime(Timer timer) {
    return ((float) ((timer.endTime.tv_sec - timer.startTime.tv_sec) \
                + (timer.endTime.tv_usec - timer.startTime.tv_usec)/1.0e6));
}
